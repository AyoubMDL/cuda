#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "kernel_launcher.h"

__global__ void reduce0(const float *input, float *output, int size) {
    extern __shared__ float sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = (index < size) ? input[index] : 0.0f;
    __syncthreads();

    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        // highly divergent warps are very inefficient, and % operator is very
        // slow
        if (tid % (2 * stride) == 0) {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}

void launch_reduce0(const float *d_input, float *d_output, int size,
                    int numBlocks, int threadsPerBlock) {
    reduce0<<<numBlocks, threadsPerBlock, threadsPerBlock * sizeof(float)>>>(
        d_input, d_output, size);
}
