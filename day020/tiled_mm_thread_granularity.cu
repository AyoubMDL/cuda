#include <hip/hip_runtime.h>
#include <iostream>

#define TILE_WIDTH 32


__global__ void tiledMatMulOptimized(const float *A, const float *B, float *C, int width) {
    __shared__ float sharedA[TILE_WIDTH][TILE_WIDTH];
    // Load two columns of B matrix
    __shared__ float sharedB[TILE_WIDTH][TILE_WIDTH * 2];

    int ty = threadIdx.y;
    int tx = threadIdx.x;
    int row = blockIdx.y * TILE_WIDTH + ty;

    int col1 = blockIdx.x * TILE_WIDTH * 2 + tx;
    int col2 = col1 + TILE_WIDTH;

    float Cvalue1 = 0;
    float Cvalue2 = 0;

    for (int t = 0; t < ceil((float)width / TILE_WIDTH); ++t) {
        // Load A tile
        int tileRow = row;
        int tileCol = t * TILE_WIDTH + tx;
        sharedA[ty][tx] = A[tileRow * width + tileCol];
        
        // Load two columns of N tile
        tileRow = t * TILE_WIDTH + ty;
        tileCol = col1;
        sharedB[ty][tx] = B[tileRow * width + tileCol];
        if (col2 < width) {
            sharedB[ty][tx + TILE_WIDTH] = B[tileRow * width + col2];
        }
        __syncthreads();

        // Dot product
        for (int k = 0; k < TILE_WIDTH; ++k) {
            Cvalue1 += sharedA[ty][k] * sharedB[k][tx];
            if (col2 < width) {
                Cvalue2 += sharedA[ty][k] * sharedB[k][tx + TILE_WIDTH];
            }
        }
        __syncthreads();
    }
    C[row * width + col1] = Cvalue1;
    if (col2 < width) {
        C[row * width + col2] = Cvalue2;
    }
}


int main() {
    const int width = 2048;
    float *A, *B, *C;

    // Allocate pinned host memory (for better transfer performance)
    hipHostMalloc((void**)&A, width * width * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void**)&B, width * width * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void**)&C, width * width * sizeof(float), hipHostMallocDefault);

    for (int i = 0; i < width; ++i) {
        for (int j = 0; j < width; ++j) {
            A[i * width + j] = 1.0f;
            B[i * width + j] = (i == j) ? 1.0f : 0.0f;  // Identity matrix
            C[i * width + j] = 0.0f;
        }
    }

    float *dA, *dB, *dC;
    hipMalloc(&dA, width * width * sizeof(float));
    hipMalloc(&dB, width * width * sizeof(float));
    hipMalloc(&dC, width * width * sizeof(float));

    hipMemcpy(dA, A, width * width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, width * width * sizeof(float), hipMemcpyHostToDevice);

    dim3 block(TILE_WIDTH, TILE_WIDTH);
    dim3 grid(
        ceil((float) width / (TILE_WIDTH * 2)), // cols
        ceil((float) width / TILE_WIDTH) // rows
    );

    // Timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    tiledMatMulOptimized<<<grid, block>>>(dA, dB, dC, width);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(C, dC, width * width * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Tiled Matmul Time with Thread granularity: " << milliseconds << " ms" << std::endl;

    // Validate result (C = A @ B = A ; B = I)
    bool valid = true;
    for (int i = 0; i < width; ++i) {
        for (int j = 0; j < width; ++j) {
            if (C[i * width + j] != A[i * width + j]) {
                valid = false;
                break;
            }
        }
    }
    std::cout << "Validation: " << (valid ? "PASSED" : "FAILED") << std::endl;

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);

    return 0;
}